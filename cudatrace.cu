#include "hip/hip_runtime.h"
/* c-ray-f - a simple raytracing filter.
 * Copyright (C) 2006 John Tsiombikas <nuclear@siggraph.org>
 *
 * You are free to use, modify and redistribute this program under the
 * terms of the GNU General Public License v2 or (at your option) later.
 * see "http://www.gnu.org/licenses/gpl.txt" for details.
 * ---------------------------------------------------------------------
 * Usage:
 *   compile:  cc -o c-ray-f c-ray-f.c -lm
 *   run:      cat scene | ./c-ray-f >foo.ppm
 *   enjoy:    display foo.ppm (with imagemagick)
 *      or:    imgview foo.ppm (on IRIX)
 * ---------------------------------------------------------------------
 * Scene file format:
 *   # sphere (many)
 *   s  x y z  rad   r g b   shininess   reflectivity
 *   # light (many)
 *   l  x y z
 *   # camera (one)
 *   c  x y z  fov   tx ty tz
 * ---------------------------------------------------------------------
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <ctype.h>
#include <errno.h>

/* find the appropriate way to define explicitly sized types */
#if (__STDC_VERSION__ >= 199900) || defined(__GLIBC__)	/* C99 or GNU libc */
#include <stdint.h>
#elif defined(__unix__) || defined(unix)
#include <sys/types.h>
#elif defined(_MSC_VER)	/* the nameless one */
typedef unsigned __int8 u_int8_t;
typedef unsigned __int32 u_int32_t;
#endif

#define cudaErrorCheck(call) { cudaAssert(call,__FILE__,__LINE__); }

void cudaAssert(const hipError_t err, const char *file, const int line)
{ 
    if( hipSuccess != err) {                                                
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        
                file, line, hipGetErrorString(err) );
        exit(1);
    } 
}

struct vec3 {
    double x, y, z;
};

struct ray {
    struct vec3 orig, dir;
};


struct reflectdata {  //STRUCT WHICH CONTAINS THE DATA FOR TRACING FURTHER REFLECTION RAYS
    struct ray r;
    double reflection;
}; 

struct material {
    struct vec3 col;	/* color */
    double spow;		/* specular power */
    double refl;		/* reflection intensity */
};

struct sphere {
    struct vec3 pos;
    double rad;
    struct material mat;
    struct sphere *next;
    int notnull;
};

struct spoint {
    struct vec3 pos, normal, vref;	/* position, normal and view reflection */
    double dist;		/* parametric distance of intersection along the ray */
};

struct camera {
    struct vec3 pos, targ;
    double fov;
};

void render1(int xsz, int ysz, u_int32_t *fb, int samples);
__global__ void render2(int xsz, int ysz, u_int32_t *fb, int samples, struct sphere *obj_list_flat_dev, int lnumdev, struct camera *camdev, struct vec3 *lightsdev, struct vec3 *uranddev, int *iranddev, int *OBJCOUNTERDEV);
__device__ struct vec3 trace(struct ray ray, int *depth, int *isReflect, struct reflectdata *RData, struct sphere *obj_list_flat, int lnumdev, struct vec3 *lightsdev, int *OBJCOUNTERDEV);
__device__ struct vec3 shade(struct sphere *obj, struct spoint *sp, int *depth, int *isReflect, struct reflectdata *Rdata, struct sphere *obj_list_flat_dev, int lnumdev, struct vec3 *lightsdev);
__device__ struct vec3 reflect(struct vec3 v, struct vec3 n);
__device__ struct vec3 cross_product(struct vec3 v1, struct vec3 v2);
__device__ struct ray get_primary_ray(int x, int y, int sample, struct camera *camdev, struct vec3 *uranddev, int *iranddev);
__device__ struct vec3 get_sample_pos(int x, int y, int sample, struct vec3 *uranddev, int *iranddev);
__device__ struct vec3 jitter(int x, int y, int s, struct vec3 *uranddev, int *iranddev);
__device__ int ray_sphere(struct sphere *sph, struct ray ray, struct spoint *sp);
void load_scene(FILE *fp);

void flatten_obj_list(struct sphere *obj_list, struct sphere *obj_list_flat, int OBJCOUNTER);
unsigned long get_msec(void);

#define MAX_LIGHTS		16				/* maximum number of lights */
#define RAY_MAG			1000.0			/* trace rays of this magnitude */
#define MAX_RAY_DEPTH	5				/* raytrace recursion limit */
#define FOV				0.78539816		/* field of view in rads (pi/4) */
#define HALF_FOV		(FOV * 0.5)
#define ERR_MARGIN		1e-6			/* an arbitrary error margin to avoid surface acne */

/* bit-shift ammount for packing each color into a 32bit u_int */
#ifdef LITTLE_ENDIAN
#define RSHIFT	16
#define BSHIFT	0
#else	/* big endian */
#define RSHIFT	0
#define BSHIFT	16
#endif	/* endianess */
#define GSHIFT	8	/* this is the same in both byte orders */

/* some helpful macros... */
#define SQ(x)		((x) * (x))
#define MAX(a, b)	((a) > (b) ? (a) : (b))
#define MIN(a, b)	((a) < (b) ? (a) : (b))
#define DOT(a, b)	((a).x * (b).x + (a).y * (b).y + (a).z * (b).z)
#define NORMALIZE(a)  do {\
    double len = sqrt(DOT(a, a));\
    (a).x /= len; (a).y /= len; (a).z /= len;\
} while(0);

/* global state */
int xres = 800;
int yres = 800;
double aspect = 1;
struct sphere *obj_list;
struct sphere *obj_list_flat;
struct vec3 lights[MAX_LIGHTS];
int lnum = 0;
int OBJCOUNTER=0;
struct camera cam;

__device__ int xresdev = 800;
__device__ int yresdev = 800;
__device__ double aspectdev = 1;

#define NRAN	1024
#define MASK	(NRAN - 1)
struct vec3 urand[NRAN];
int irand[NRAN];

const char *usage = {
    "Usage: c-ray-f [options]\n"
        "  Reads a scene file from stdin, writes the image to stdout, and stats to stderr.\n\n"
        "Options:\n"
        "  -s WxH     where W is the width and H the height of the image\n"
        "  -r <rays>  shoot <rays> rays per pixel (antialiasing)\n"
        "  -i <file>  read from <file> instead of stdin\n"
        "  -o <file>  write to <file> instead of stdout\n"
        "  -h         this help screen\n\n"
};



int main(int argc, char **argv) {
    int i, j;
    unsigned long rend_time, start_time;
    u_int32_t *pixels;
    int rays_per_pixel = 1;
    FILE *infile = stdin, *outfile = stdout;

    for(i=1; i<argc; i++) {
        if(argv[i][0] == '-' && argv[i][2] == 0) {
            char *sep;
            switch(argv[i][1]) {
                case 's':
                    if(!isdigit(argv[++i][0]) || !(sep = strchr(argv[i], 'x')) || !isdigit(*(sep + 1))) {
                        fputs("-s must be followed by something like \"640x480\"\n", stderr);
                        return EXIT_FAILURE;
                    }
                    xres = atoi(argv[i]);
                    yres = atoi(sep + 1);
                    aspect = (double)xres / (double)yres;
                    break;

                case 'i':
                    if(!(infile = fopen(argv[++i], "r"))) {
                        fprintf(stderr, "failed to open input file %s: %s\n", argv[i], strerror(errno));
                        return EXIT_FAILURE;
                    }
                    break;

                case 'o':
                    if(!(outfile = fopen(argv[++i], "w"))) {
                        fprintf(stderr, "failed to open output file %s: %s\n", argv[i], strerror(errno));
                        return EXIT_FAILURE;
                    }
                    break;

                case 'r':
                    if(!isdigit(argv[++i][0])) {
                        fputs("-r must be followed by a number (rays per pixel)\n", stderr);
                        return EXIT_FAILURE;
                    }
                    rays_per_pixel = atoi(argv[i]);
                    break;

                case 'h':
                    fputs(usage, stdout);
                    return 0;

                default:
                    fprintf(stderr, "unrecognized argument: %s\n", argv[i]);
                    fputs(usage, stderr);
                    return EXIT_FAILURE;
            }
        } else {
            fprintf(stderr, "unrecognized argument: %s\n", argv[i]);
            fputs(usage, stderr);
            return EXIT_FAILURE;
        }
    }

    if(!(pixels = (u_int32_t *)malloc(xres * yres * sizeof *pixels))) {
        perror("pixel buffer allocation failed");
        return EXIT_FAILURE;
    }
    load_scene(infile);

    obj_list_flat = (struct sphere *)malloc(sizeof(struct sphere)*OBJCOUNTER+1);  //plus one for the null element at the end
    flatten_obj_list(obj_list,obj_list_flat,OBJCOUNTER);

    /* initialize the random number tables for the jitter */
    for(i=0; i<NRAN; i++) urand[i].x = (double)rand() / RAND_MAX - 0.5;
    for(i=0; i<NRAN; i++) urand[i].y = (double)rand() / RAND_MAX - 0.5;
    for(i=0; i<NRAN; i++) irand[i] = (int)(NRAN * ((double)rand() / RAND_MAX));

    start_time = get_msec();
    render1(xres, yres, pixels, rays_per_pixel);
    rend_time = get_msec() - start_time;

    /* output statistics to stderr */
    fprintf(stderr, "Rendering took: %lu seconds (%lu milliseconds)\n", rend_time / 1000, rend_time);

    // output the image 
    fprintf(outfile, "P6\n%d %d\n255\n", xres, yres);
    for(j=0; j<yres; j++) {
        for(i=0; i<xres; i++) {
            fputc((pixels[i + (xres * j)] >> RSHIFT) & 0xff, outfile);
            fputc((pixels[i + (xres * j)] >> GSHIFT) & 0xff, outfile);
            fputc((pixels[i + (xres * j)] >> BSHIFT) & 0xff, outfile);
        }
    }
    fflush(outfile);

    if(infile != stdin) fclose(infile);
    if(outfile != stdout) fclose(outfile);
    return 0;
}


void render1(int xsz, int ysz, u_int32_t *host_fb, int samples)
{
    dim3 threads_per_block(16, 16);

    int whole_blocks_x = xsz/threads_per_block.x;
    int whole_blocks_y = ysz/threads_per_block.y;

    int remainder_threads_x = xsz % threads_per_block.x;
    int remainder_threads_y = ysz % threads_per_block.y;

    int extra_block_x = 0;
    int extra_block_y = 0;

    if (remainder_threads_x > 0) {
        extra_block_x = 1;
    }

    if (remainder_threads_y > 0) {
        extra_block_y = 1;
    }

    int num_blocks_x = whole_blocks_x + extra_block_x;
    int num_blocks_y = whole_blocks_y + extra_block_y;

    dim3 num_blocks(num_blocks_x, num_blocks_y);
    
    size_t arr_size = xsz * ysz * sizeof(u_int32_t);

    u_int32_t *device_fb = 0;
    cudaErrorCheck(hipMalloc((void **)&device_fb, arr_size));
    cudaErrorCheck(hipMemcpy(device_fb, host_fb, arr_size, hipMemcpyHostToDevice));

    struct sphere *obj_list_flat_dev;

    //create obj_list_flat_dev array size of objCounter
    cudaErrorCheck(hipMalloc((void **)&obj_list_flat_dev, (sizeof(struct sphere)*(OBJCOUNTER+1))));
    cudaErrorCheck(hipMemcpy(obj_list_flat_dev, obj_list_flat, (sizeof(struct sphere)*OBJCOUNTER+1), hipMemcpyHostToDevice)); //copying over flat sphere array to obj_listdevflat

    int *OBJCOUNTERDEV = 0;
    cudaErrorCheck(hipMalloc((void**)&OBJCOUNTERDEV, sizeof(int)));
    cudaErrorCheck( hipMemcpy(OBJCOUNTERDEV, &OBJCOUNTER, sizeof(int), hipMemcpyHostToDevice) );

    //lights and camera and whatnot
    int lnumdev = 0;

    struct camera *camdev = 0;
    cudaErrorCheck(hipMalloc((void **)&camdev, sizeof(struct camera)) );
    cudaErrorCheck(hipMemcpy(camdev, &cam, sizeof(struct camera), hipMemcpyHostToDevice));

    struct vec3 *lightsdev = 0;
    cudaErrorCheck(hipMalloc((void **)&lightsdev, MAX_LIGHTS*sizeof(struct vec3)) );
    cudaErrorCheck(hipMemcpy(lightsdev, lights, MAX_LIGHTS*sizeof(struct vec3), hipMemcpyHostToDevice));

    lnumdev = lnum; //remember to pass lnumdev into render2!
    //camdev = cam;   //remember to pass camdev into render2!

    //urand and whatnot
    struct vec3 *uranddev = 0;
    cudaErrorCheck(hipMalloc((void **)&uranddev, NRAN*sizeof(struct vec3)) );
    cudaErrorCheck(hipMemcpy(uranddev, urand, sizeof(struct vec3) * NRAN, hipMemcpyHostToDevice)); //remember to pass all of these into render2!!

    //irand and whatnot
    int *iranddev = 0;
    cudaErrorCheck(hipMalloc((void **)&iranddev, NRAN*sizeof(int)) );
    cudaErrorCheck(hipMemcpy(iranddev, irand, sizeof(int) * NRAN, hipMemcpyHostToDevice)); //remember to pass all of these into render2!!

    // KERNEL CALL!
    //render2<<<num_blocks, threads_per_block>>>(xsz, ysz, device_fb, samples, obj_list_flat_dev, lnumdev, camdev, lightsdev, uranddev, iranddev, OBJCOUNTERDEV);
    render2<<<1,1>>>(xsz, ysz, device_fb, samples, obj_list_flat_dev, lnumdev, camdev, lightsdev, uranddev, iranddev, OBJCOUNTERDEV);
    hipPeekAtLastError(); // Checks for launch error
    cudaErrorCheck( hipDeviceSynchronize() );

    //In all seriousness, all of the cores should now be operating on the ray tracing, if things are working correctly 
    //once done, copy contents of device array to host array  

    cudaErrorCheck(hipMemcpy(lights, lightsdev, sizeof(struct vec3) * MAX_LIGHTS, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipMemcpy(&cam, camdev, sizeof(struct camera), hipMemcpyDeviceToHost));
    cudaErrorCheck(hipMemcpy(host_fb, device_fb, arr_size, hipMemcpyDeviceToHost));
    //printf("cam.pos.x: %f\n", cam.pos.x);

    free(obj_list_flat);
    cudaErrorCheck( hipFree(lightsdev) );
    cudaErrorCheck( hipFree(uranddev) );
    cudaErrorCheck( hipFree(iranddev) );
    cudaErrorCheck( hipFree(device_fb) );
    cudaErrorCheck( hipFree(obj_list_flat_dev) );
}   

/* render a frame of xsz/ysz dimensions into the provided framebuffer */
__global__ void render2(int xsz, int ysz, u_int32_t *fb, int samples, struct sphere *obj_list_flat_dev, int lnumdev, struct camera *camdev, struct vec3 *lightsdev, struct vec3 *uranddev, int *iranddev, int *OBJCOUNTERDEV) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int grid_width = gridDim.x * blockDim.x;
    int index = j * grid_width + i;

    int s;

    if ((i < xsz) && (j < ysz)) {
        double rcp_samples = 1.0 / (double)samples;

        /* for each subpixel, trace a ray through the scene, accumulate the
         * colors of the subpixels of each pixel, then pack the color and
         * put it into the framebuffer.
         * XXX: assumes contiguous scanlines with NO padding, and 32bit pixels.
         */


        int isReflect[1]; //WHETHER OR NOT RAY TRACED WILL NEED A REFLECTION RAY AS WELL
        isReflect[0] = 0;
        int depth[1];
        depth[0]=0;
        struct reflectdata RData[1]; //ARRAY WHICH CONTAINS REFLECT DATA STRUCT TO BE PASSED ON TO TRACE FUNCTION

        double r, g, b;
        r = g = b = 0.0;

        for(s=0; s<samples; s++) {

            struct vec3 col = trace(get_primary_ray(i, j, s, camdev, uranddev, iranddev), depth, isReflect, RData, obj_list_flat_dev, lnumdev, lightsdev, OBJCOUNTERDEV);

            while (isReflect[0])        //while there are still reflection rays to trace
            {
                struct vec3 rcol;    //holds the output of the reflection ray calculcation
                rcol = trace(RData->r, depth, isReflect, RData, obj_list_flat_dev, lnumdev, lightsdev, OBJCOUNTERDEV);    //trace a reflection ray
                col.x += rcol.x * RData->reflection;       //I really am unsure about the usage of pointers here..
                col.y += rcol.y * RData->reflection;
                col.z += rcol.z * RData->reflection;
            }   
            depth[0] = 0;


            r += col.x;
            g += col.y;
            b += col.z;
        }

        r = r * rcp_samples;
        g = g * rcp_samples;
        b = b * rcp_samples;

        fb[index] =        ((u_int32_t)(MIN(r, 1.0) * 255.0) & 0xff) << RSHIFT |
                           ((u_int32_t)(MIN(g, 1.0) * 255.0) & 0xff) << GSHIFT |
                           ((u_int32_t)(MIN(b, 1.0) * 255.0) & 0xff) << BSHIFT;

    }
}

/* trace a ray throught the scene recursively (the recursion happens through
 * shade() to calculate reflection rays if necessary).
 */
__device__ struct vec3 trace(struct ray ray, int *depth, int *isReflect, struct reflectdata *RData, struct sphere *obj_list_flat_dev, int lnumdev, struct vec3 *lightsdev, int *OBJCOUNTERDEV) {
    struct vec3 col;
    struct spoint sp, nearest_sp;
    struct sphere nearest_obj;
    nearest_obj.notnull = 0;
    //	struct sphere *iter = obj_list->next;

    int iterincr = 0;
    struct sphere *iter = obj_list_flat_dev;

    /* if we reached the recursion limit, bail out */

    if(depth[0] >= MAX_RAY_DEPTH) {
        col.x = col.y = col.z = 0.0;
        isReflect[0] = 0;
        return col;
    }

    /* find the nearest intersection ... */
    while(iter->notnull) {
        if(ray_sphere(iter, ray, &sp)) {
            if(!(nearest_obj.notnull) || sp.dist < nearest_sp.dist) {
                nearest_obj = *iter;
                nearest_sp = sp;
            }
        }
        iterincr++;
        iter = &(obj_list_flat_dev[iterincr]);
        //iter = iter->next;
    }

    /* and perform shading calculations as needed by calling shade() */
    if(nearest_obj.notnull) {
        col = shade(&nearest_obj, &nearest_sp, depth, isReflect, RData, obj_list_flat_dev, lnumdev, lightsdev);
    } else {
        col.x = col.y = col.z = 0.0;
        isReflect[0]=0;
    }

    return col;
}

/* Calculates direct illumination with the phong reflectance model.
 * Also handles reflections by calling trace again, if necessary.
 */
__device__ struct vec3 shade(struct sphere *obj, struct spoint *sp, int *depth, int *isReflect, struct reflectdata *Rdata, struct sphere *obj_list_flat_dev, int lnumdev, struct vec3 *lightsdev) {
    int i;
    struct vec3 col = {0, 0, 0};

    /* for all lights ... */
    for(i=0; i<lnumdev; i++) {
        double ispec, idiff;
        struct vec3 ldir;
        struct ray shadow_ray;
        //		struct sphere *iter = obj_list->next;

        int iterincr = 0;

        struct sphere *iter = obj_list_flat_dev;
        int in_shadow = 0;

        ldir.x = lightsdev[i].x - sp->pos.x;
        ldir.y = lightsdev[i].y - sp->pos.y;
        ldir.z = lightsdev[i].z - sp->pos.z;

        shadow_ray.orig = sp->pos;
        shadow_ray.dir = ldir;

        /* shoot shadow rays to determine if we have a line of sight with the light */
        while(iter->notnull) {
            if(ray_sphere(iter, shadow_ray, 0)) {
                in_shadow = 1;
                break;
            }
            iterincr++;
            iter = &(obj_list_flat_dev[iterincr]);
            //	iter = iter->next;
        }

        /* and if we're not in shadow, calculate direct illumination with the phong model. */
        if(!in_shadow) {
            NORMALIZE(ldir);

            idiff = MAX(DOT(sp->normal, ldir), 0.0);
            ispec = obj->mat.spow > 0.0 ? pow(MAX(DOT(sp->vref, ldir), 0.0), obj->mat.spow) : 0.0;

            col.x += idiff * obj->mat.col.x + ispec;
            col.y += idiff * obj->mat.col.y + ispec;
            col.z += idiff * obj->mat.col.z + ispec;
        }
    }

    /* Also, if the object is reflective, spawn a reflection ray, and call trace()
     * to calculate the light arriving from the mirror direction.
     */
    if(obj->mat.refl > 0.0) {

        isReflect[0] = 1;    //set isReflect to affirmative 

        Rdata->r.orig = sp->pos;     //SET VALUES OF REFLECTIONDATA STRUCT
        Rdata->r.dir = sp->vref;
        Rdata->r.dir.x *= RAY_MAG;
        Rdata->r.dir.y *= RAY_MAG;
        Rdata->r.dir.z *= RAY_MAG;
        depth[0] = *depth + 1;
        Rdata->reflection = obj->mat.refl;
    }
    else {
        isReflect[0] = 0;
    }

    return col;
}

/* calculate reflection vector */
struct vec3 reflect(struct vec3 v, struct vec3 n) {
    struct vec3 res;
    double dot = v.x * n.x + v.y * n.y + v.z * n.z;
    res.x = -(2.0 * dot * n.x - v.x);
    res.y = -(2.0 * dot * n.y - v.y);
    res.z = -(2.0 * dot * n.z - v.z);
    return res;
}

struct vec3 cross_product(struct vec3 v1, struct vec3 v2) {
    struct vec3 res;
    res.x = v1.y * v2.z - v1.z * v2.y;
    res.y = v1.z * v2.x - v1.x * v2.z;
    res.z = v1.x * v2.y - v1.y * v2.x;
    return res;
}

/* determine the primary ray corresponding to the specified pixel (x, y) */
__device__ struct ray get_primary_ray(int x, int y, int sample, struct camera *camdev, struct vec3 *uranddev, int *iranddev) {
    struct ray ray;
    float m[3][3];
    struct vec3 i, j = {0, 1, 0}, k, dir, orig, foo;

    k.x = camdev->targ.x - camdev->pos.x;
    k.y = camdev->targ.y - camdev->pos.y;
    k.z = camdev->targ.z - camdev->pos.z;
    NORMALIZE(k);

    i = cross_product(j, k);
    j = cross_product(k, i);
    m[0][0] = i.x; m[0][1] = j.x; m[0][2] = k.x;
    m[1][0] = i.y; m[1][1] = j.y; m[1][2] = k.y;
    m[2][0] = i.z; m[2][1] = j.z; m[2][2] = k.z;

    ray.orig.x = ray.orig.y = ray.orig.z = 0.0;
    ray.dir = get_sample_pos(x, y, sample, uranddev, iranddev);
    ray.dir.z = 1.0 / HALF_FOV;
    ray.dir.x *= RAY_MAG;
    ray.dir.y *= RAY_MAG;
    ray.dir.z *= RAY_MAG;

    dir.x = ray.dir.x + ray.orig.x;
    dir.y = ray.dir.y + ray.orig.y;
    dir.z = ray.dir.z + ray.orig.z;
    foo.x = dir.x * m[0][0] + dir.y * m[0][1] + dir.z * m[0][2];
    foo.y = dir.x * m[1][0] + dir.y * m[1][1] + dir.z * m[1][2];
    foo.z = dir.x * m[2][0] + dir.y * m[2][1] + dir.z * m[2][2];

    orig.x = ray.orig.x * m[0][0] + ray.orig.y * m[0][1] + ray.orig.z * m[0][2] + camdev->pos.x;
    orig.y = ray.orig.x * m[1][0] + ray.orig.y * m[1][1] + ray.orig.z * m[1][2] + camdev->pos.y;
    orig.z = ray.orig.x * m[2][0] + ray.orig.y * m[2][1] + ray.orig.z * m[2][2] + camdev->pos.z;

    ray.orig = orig;
    ray.dir.x = foo.x + orig.x;
    ray.dir.y = foo.y + orig.y;
    ray.dir.z = foo.z + orig.z;

    return ray;
}


__device__ struct vec3 get_sample_pos(int x, int y, int sample, struct vec3 *uranddev, int *iranddev) {
    struct vec3 pt;
    double xsz = 2.0, ysz = xresdev / aspectdev;
    /*static */ double sf = 0.0;

    if(sf == 0.0) {
        sf = 2.0 / (double)xresdev;
    }

    pt.x = ((double)x / (double)xresdev) - 0.5;
    pt.y = -(((double)y / (double)yresdev) - 0.65) / aspectdev;

    if(sample) {
        struct vec3 jt = jitter(x, y, sample, uranddev, iranddev);
        pt.x += jt.x * sf;
        pt.y += jt.y * sf / aspectdev;
    }
    return pt;
}

/* jitter function taken from Graphics Gems I. */
__device__ struct vec3 jitter(int x, int y, int s, struct vec3 *uranddev, int *iranddev) {
    struct vec3 pt;
    pt.x = uranddev[(x + (y << 2) + iranddev[(x + s) & MASK]) & MASK].x;
    pt.y = uranddev[(y + (x << 2) + iranddev[(y + s) & MASK]) & MASK].y;
    return pt;
}

/* Calculate ray-sphere intersection, and return {1, 0} to signify hit or no hit.
 * Also the surface point parameters like position, normal, etc are returned through
 * the sp pointer if it is not NULL.
 */
__device__ int ray_sphere(struct sphere *sph, struct ray ray, struct spoint *sp) {
    double a, b, c, d, sqrt_d, t1, t2;

    a = SQ(ray.dir.x) + SQ(ray.dir.y) + SQ(ray.dir.z);
    b = 2.0 * ray.dir.x * (ray.orig.x - sph->pos.x) +
        2.0 * ray.dir.y * (ray.orig.y - sph->pos.y) +
        2.0 * ray.dir.z * (ray.orig.z - sph->pos.z);
    c = SQ(sph->pos.x) + SQ(sph->pos.y) + SQ(sph->pos.z) +
        SQ(ray.orig.x) + SQ(ray.orig.y) + SQ(ray.orig.z) +
        2.0 * (-sph->pos.x * ray.orig.x - sph->pos.y * ray.orig.y - sph->pos.z * ray.orig.z) - SQ(sph->rad);

    if((d = SQ(b) - 4.0 * a * c) < 0.0) return 0;

    sqrt_d = sqrt(d);
    t1 = (-b + sqrt_d) / (2.0 * a);
    t2 = (-b - sqrt_d) / (2.0 * a);

    if((t1 < ERR_MARGIN && t2 < ERR_MARGIN) || (t1 > 1.0 && t2 > 1.0)) return 0;

    if(sp) {
        if(t1 < ERR_MARGIN) t1 = t2;
        if(t2 < ERR_MARGIN) t2 = t1;
        sp->dist = t1 < t2 ? t1 : t2;

        sp->pos.x = ray.orig.x + ray.dir.x * sp->dist;
        sp->pos.y = ray.orig.y + ray.dir.y * sp->dist;
        sp->pos.z = ray.orig.z + ray.dir.z * sp->dist;

        sp->normal.x = (sp->pos.x - sph->pos.x) / sph->rad;
        sp->normal.y = (sp->pos.y - sph->pos.y) / sph->rad;
        sp->normal.z = (sp->pos.z - sph->pos.z) / sph->rad;

        sp->vref = reflect(ray.dir, sp->normal);
        NORMALIZE(sp->vref);
    }
    return 1;
}

/* Load the scene from an extremely simple scene description file */
#define DELIM	" \t\n"
void load_scene(FILE *fp) {
    OBJCOUNTER = 0;
    char line[256], *ptr, type;

    obj_list = (struct sphere *)malloc(sizeof(struct sphere));
    obj_list->next = 0;

    while((ptr = fgets(line, 256, fp))) {
        int i;
        struct vec3 pos, col;
        double rad, spow, refl;

        while(*ptr == ' ' || *ptr == '\t') ptr++;
        if(*ptr == '#' || *ptr == '\n') continue;

        if(!(ptr = strtok(line, DELIM))) continue;
        type = *ptr;

        for(i=0; i<3; i++) {
            if(!(ptr = strtok(0, DELIM))) break;
            *((double*)&pos.x + i) = atof(ptr);
        }

        if(type == 'l') {
            lights[lnum++] = pos;
            continue;
        }

        if(!(ptr = strtok(0, DELIM))) continue;
        rad = atof(ptr);

        for(i=0; i<3; i++) {
            if(!(ptr = strtok(0, DELIM))) break;
            *((double*)&col.x + i) = atof(ptr);
        }

        if(type == 'c') {
            cam.pos = pos;
            cam.targ = col;
            cam.fov = rad;
            continue;
        }

        if(!(ptr = strtok(0, DELIM))) continue;
        spow = atof(ptr);

        if(!(ptr = strtok(0, DELIM))) continue;
        refl = atof(ptr);

        if(type == 's') {
            struct sphere *sph = (struct sphere *)malloc(sizeof *sph);
            sph->next = obj_list->next;
            obj_list->next = sph;

            sph->pos = pos;
            sph->rad = rad;
            sph->mat.col = col;
            sph->mat.spow = spow;
            sph->mat.refl = refl;
            OBJCOUNTER++;
        } else {
            fprintf(stderr, "unknown type: %c\n", type);
        }
    }
}



void flatten_obj_list(struct sphere *obj_list, struct sphere *obj_list_flat, int OBJCOUNTER) {


    struct sphere *sphere = obj_list;
    sphere = sphere->next;

    int j;
    for (j = 0; j < OBJCOUNTER; j++) {
        obj_list_flat[j] = *sphere;
        obj_list_flat[j].notnull = 1;
        sphere = sphere->next;

    }
    obj_list_flat[j].notnull = 0;





}


/* provide a millisecond-resolution timer for each system */
#if defined(__unix__) || defined(unix) || defined(__MACH__)
#include <time.h>
#include <sys/time.h>
unsigned long get_msec(void) {
    static struct timeval timeval, first_timeval;

    gettimeofday(&timeval, 0);
    if(first_timeval.tv_sec == 0) {
        first_timeval = timeval;
        return 0;
    }
    return (timeval.tv_sec - first_timeval.tv_sec) * 1000 + (timeval.tv_usec - first_timeval.tv_usec) / 1000;
}
#elif defined(__WIN32__) || defined(WIN32)
#include <windows.h>
unsigned long get_msec(void) {
    return GetTickCount();
}
#else
#error "I don't know how to measure time on your platform"
#endif


